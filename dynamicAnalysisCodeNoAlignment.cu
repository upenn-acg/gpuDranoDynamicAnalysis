
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

typedef uint64_t int64;
#define warpSizeD 32

/* =====================================================================================*/
/** Given an integer representing a 32-entry array of bits, return the nth bit of the
 * array as either 0 or 1, and a number of bits to shift.
 */
__device__ int getNthBit(unsigned int bitArray, int nth){
  return 1 & (bitArray >> nth);
}
/* =====================================================================================*/
/**
 * Function to count the number of unique cache lines needed for each load or store.
 * Prints information about location of load/store and number of cache lines needed.
 * @param: address of load/store we want to compute for.
 * @param: name of module where this load/store resides.
 * @param: function name where this load/store resides.
 * @param: either "load" or "store"
 * @param: a unique integer given to each invocation to this function to differentiate
 *         different dynamic instructions.
 */
__device__ void countCacheLines(void* addressP, char* moduleName, char* functionName,
                                char* loadOrStore, int lineNum, int columnNum,
                                int dynamicId, int typeSize){
  // Not a global memory address.
  if(1 != __isGlobal(addressP))
    return;

  /* Not all threads may be active in this function. We use the cuda ballot() function to
     figure out which treads are currently active. */
  int activeThreads =__ballot(1);
  // Our address will be used as a "None" value. Since we know it will not cause
  // problems when counting unique values. Notice this will only matter for the
  // reduce thread.
  int64 address = (int64) addressP;

  // Array to hold the addresses of all the threads. Twice as big as the warp
  // since we want the starting (min adress) and ending address (max adress) for every
  // read, that is, all the bytes a single thread is accessing.
  int64 addrArray[2 * warpSizeD];

  // Thread to gather values across threads.
  int reduceThread = -1;
  for(int i = 0; i < warpSizeD; i++)
    if(getNthBit(activeThreads, i) == 1){
      reduceThread = i;
      break;
    }

  // Shuffle values from all threads to our addrArray. Shuffling is undefined if we ask an
  // unactive thread. So we only query active threads.
  for(int i = 0; i < warpSizeD; i++){
    if(getNthBit(activeThreads, i) == 0)
      addrArray[i * 2] = address;
    else{
      // Break our shuffle into higher and lower order bits.
      int hob = (int)(address >> 32);
      int lob = 0xFFFFFFFF & address;
      hob = __shfl(hob, i);
      lob = __shfl(lob, i);
      addrArray[i * 2] = (((int64) hob) << 32) | (int64) lob;
    }
  }
  // We are computing based on warps, but thread id's go past 32. So we must modulo 
  // around.
  if(reduceThread == (threadIdx.x % warpSizeD)){
    // Number of unique cache lines.
    int count = 1;

    // Figure out upper byte accesses per thread.
    for(int i = 0; i < 2 * warpSizeD; i += 2){
      addrArray[i + 1] = addrArray[i] + typeSize - 1;
    }

    int64 min = addrArray[reduceThread];
    // Ignore alignment issues by finding minimum element and subtracting from all others.
    for(int i = reduceThread + 1; i < 2 * warpSizeD; i++){
      if(min > addrArray[reduceThread])
        min = addrArray[reduceThread];
    }

    for(int i = 0; i < 2 * warpSizeD; i ++){
      addrArray[i] = (addrArray[i] - min) >> 7;
    }

    int64 myNone = addrArray[reduceThread];

    // Count unique elements.
    for(int i = reduceThread + 1; i < 2 * warpSizeD; i++)
      if(addrArray[i] != myNone){       // Skip inactive threads.
        int64 current = addrArray[i];
        count++;
        // Iterate through rest of addrArray "none-ing out" entries that match current.
        for(int j = i + 1; j < 2 * warpSizeD; j++)
          if(addrArray[j] == current)
            addrArray[j] = myNone;
      }

    // 'DA__' is needed so we have a unique identifier to grep from the program's
    // output.
    char* str = "DA__\t%s\t%s\t%d\t%s\t%d\t%d\t%d\n";
    printf(str, moduleName, functionName, dynamicId, loadOrStore,
           lineNum, columnNum, count);
  }

  return;
}
/* =====================================================================================*/